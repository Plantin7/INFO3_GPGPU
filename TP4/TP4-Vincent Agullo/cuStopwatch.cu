#include <hip/hip_runtime.h>


class cuStopwatch{
    // todo: add your internal data structure, all in private
	private:
		hipEvent_t startEvent;
		hipEvent_t stopEvent;
		bool isStartEventStarted;

    public:
        cuStopwatch();
        ~cuStopwatch();
        void start();
        float stop();
};

cuStopwatch::cuStopwatch(){
    // todo: constructor
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    isStartEventStarted = false;
}

cuStopwatch::~cuStopwatch(){
    // todo: destructor
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

void cuStopwatch::start(){
    // todo: start the stopwatch, and ignore double start
    if(!isStartEventStarted){
    	hipEventRecord(startEvent);
    	isStartEventStarted = true;
    }
}

float cuStopwatch::stop(){
    // todo: stop the stopwatch and return elapsed time, ignore invalid stops (e.g. stop when not yet started or double stop)
    if(!isStartEventStarted){
    	return -1.0;
    }
    
    hipEventSynchronize(startEvent);
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);

	float ms;
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    return ms;
}